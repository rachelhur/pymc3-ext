
#include <hip/hip_runtime.h>
#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "transit_op.h"

using namespace tensorflow;

using GPUDevice = Eigen::GpuDevice;

// Define the CUDA kernel.
template <typename T>
__global__
void TransitCudaKernel(int                            grid_size,
                       const T*  __restrict__   const x,
                       const T*  __restrict__   const grid,
                       int                            size,
                       const int*  __restrict__ const indmin,
                       const int*  __restrict__ const indmax,
                       const T*  __restrict__   const z,
                       const T*  __restrict__   const r,
                       T*  __restrict__               delta)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < size; i += stride) {
    delta[i] = transit::compute_delta<T>(grid_size, x, grid, indmin[i], indmax[i], z[i], r[i]);
  }
}

template <typename T>
void TransitFunctor<GPUDevice, T>::operator()(
    const GPUDevice& d, int grid_size, const T* const x, const T* const grid,
    int size, const int* const indmin, const int* const indmax, const T* const z, const T* const r, T* delta)
{
  CudaLaunchConfig config = GetCudaLaunchConfig(size, d);
  TransitCudaKernel<T>
      <<<config.block_count, config.thread_per_block, 0, d.stream()>>>(grid_size, x, grid, size, indmin, indmax, z, r, delta);
}

template struct TransitFunctor<GPUDevice, float>;
template struct TransitFunctor<GPUDevice, double>;

#endif  // GOOGLE_CUDA